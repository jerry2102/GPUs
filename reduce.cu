#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <hip/hip_cooperative_groups.h>
#include <iostream>
#include <stdio.h>
#include <algorithm>
#include <numeric>
#include <hip/hip_vector_types.h>
using namespace cooperative_groups;

#define FULL_MASK 0xffffffff

/*---------------------------- reduce sum kernel for a single thread block -----------------------*/
__global__ void reduce_sum(int* temp) {
    thread_block block = this_thread_block();
    int lane = block.thread_rank();
    
    int length = block.size();
    int middle = block.size() / 2;
    int middle_length = middle + (length & 1);
    for (; middle > 0; ) {
        if (lane < middle) {
            temp[lane] += temp[lane + middle_length];
        }
        length = middle_length;
        middle = length / 2;
        middle_length = middle + (length & 1);
    }
}

__global__ void reduce_sum_by_warp_level_primitives(int* temp, int32_t count) {
    unsigned mask = __ballot_sync(FULL_MASK, threadIdx.x < count);
    if (threadIdx.x < count) {
        int val = temp[threadIdx.x];
        // clear temp[0], using it for storing result
        if (threadIdx.x == 0) temp[0] = 0;
        for (int offset = 16; offset > 0; offset /=2) {
            val += __shfl_down_sync(mask, val, offset);
        }
            
        if (threadIdx.x % 32 == 0) {
            atomicAdd(temp, val);
        }
    }
}

__global__ void reduce_sum_by_active_warps(int* temp, int32_t count) {
    if (threadIdx.x < count) {
        // using __activemask is incorrect as is would result in partial sums instead of total sum
        // tyhe CUDA execution model does not guarantee that all threads taking the branch together will 
        // execute the __activemask() together.
        // Implicit lock-step execution is not guaranteed
        uint32_t mask = __activemask();
        int val = temp[threadIdx.x];
        if (threadIdx.x == 0) temp[0] = 0;
        for (int offset = 16; offset > 0; offset /= 2) {
            val += __shfl_down_sync(mask, val, offset);
        }

        if (threadIdx.x % 32 == 0) {
            atomicAdd(temp, val);
        }
    }

}

__global__ void reduce_sum_using_manual_syncwarp(int* temp, int32_t count) {
    uint32_t tid = threadIdx.x;

    __shared__ int32_t shmem[32];
    shmem[tid] = temp[tid];
    __syncwarp();

    int v = shmem[tid];
    v += shmem[tid+16]; __syncwarp();
    shmem[tid] = v;     __syncwarp();
    v += shmem[tid+8];  __syncwarp();
    shmem[tid] = v;     __syncwarp();
    v += shmem[tid+4];  __syncwarp();
    shmem[tid] = v;     __syncwarp();
    v += shmem[tid+2];  __syncwarp();
    shmem[tid] = v;     __syncwarp();
    v += shmem[tid+1];  __syncwarp();
    shmem[tid] = v;

//    shmem[tid] += shmem[tid + 16]; __syncwarp();
//    shmem[tid] += shmem[tid + 8]; __syncwarp();
//    shmem[tid] += shmem[tid + 4]; __syncwarp();
//    shmem[tid] += shmem[tid + 2]; __syncwarp();
//    shmem[tid] += shmem[tid + 1]; __syncwarp();

    if (tid == 0) temp[0] = shmem[0];

}

__global__ void shuffle_values(int* temp, int32_t count) {
    float val = threadIdx.x;

    if (threadIdx.x % 32 <16) {
        val = __shfl_xor_sync(0xFFFFFFFF, val, 16);
    }
    else {
        val = __shfl_xor_sync(0xFFFFFFFF, val, 16);
    }
    printf("thread: %d, swpped: %f\n", threadIdx.x, val);
}

void test_reduce_sum() {
    int datas[] = {
        1,2,3,4,5,6,7,8,
        9,10,11,12,13,14,15,16,
        17,18,19,20,21,22,23,24,
        25,26,27,28,29,30,31,32,
        33,34,35,36,37,38,39,40
    };
    int* gdatas = nullptr;
    hipMalloc((void**)&gdatas, sizeof(datas));
    hipMemcpy((char*)gdatas, (char*)datas, sizeof(datas), hipMemcpyHostToDevice);
    //reduce_sum<<<1, 33>>>(gdatas);
    //reduce_sum_by_warp_level_primitives<<<1, 33>>>(gdatas, 33);
    //reduce_sum_by_active_warps<<<1, 40>>>(gdatas, 40);
    //reduce_sum_using_manual_syncwarp<<<1, 32>>>(gdatas, 32);
    shuffle_values<<<1, 32>>>(gdatas, 32);
    int* result = (int*)malloc(sizeof(datas));
    hipMemcpy((char*)result, (char*)gdatas, sizeof(datas), hipMemcpyDeviceToHost);
    std::cout << result[0] << ", " << result[1] << std::endl;
    hipFree(gdatas);
}
/*================================================================================================*/

/*------------------------------- reduce sum using thread block ----------------------------------*/
// 
__device__ int reduce_sum(thread_group g, int* temp, int val) {
    int lane = g.thread_rank();
    int length = g.size();
    int middle = length / 2;
    int middle_length = middle + (length & 1);
    for (/**/; middle > 0; /**/) {
        temp[lane] = val;
        g.sync();
        if (lane < middle) {
            val += temp[lane + middle_length];
        }
        length = middle_length;
        middle = length / 2;
        middle_length = middle + (length & 1);
        g.sync();
    }
    return val;
}

// each thread calculate for 
__device__ int thread_sum(int* input, int n) {
    int sum = 0;
    
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    for(; idx < (n + 3) / 4; idx += blockDim.x * gridDim.x) {
        int4 in = ((int4*)input)[idx];
        sum += in.x + in.y + in.z + in.w;
    }
    // for the last thread of the last block
    if (idx == (n + 3) / 4) {
        for (int t = 4 * idx; t < n; ++t) {
            sum += input[t];
        }
    }

    return sum;
}

__global__ void sum_kernel_block(int* sum, int* input, int n) {
    // each kernel calculate 4 consecutive items
    int single_thread_sum = thread_sum(input, n);
    extern __shared__ int temp[];
    auto g = this_thread_block();
    int block_sum = reduce_sum(g, temp, single_thread_sum);

    if (g.thread_rank() == 0) {
        atomicAdd(sum, block_sum);
    }
}

__global__ void sum_kernel_by_tile32(int* sum, int* input, int n) {
    int single_thread_sum = thread_sum(input, n);
    extern __shared__ int temp[];
    auto g = this_thread_block();
    int tile_idx = g.thread_rank() / 32;
    int* tile_temp = &temp[32 * tile_idx];
    auto tile32 = tiled_partition(g, 32);
    
    int tile_sum = reduce_sum(tile32, tile_temp, single_thread_sum);
    
    if (tile32.thread_rank() == 0) atomicAdd(sum, tile_sum);

}

template<int tilesize> 
__device__ int reduce_sum_tile_shfl(thread_block_tile<tilesize> g, int val) {
    for (int i = g.size() / 2; i > 0; i /= 2) {
        val += g.shfl_down(val, i);
    }
    return val;
}

template<int tilesize> 
__global__ void sum_kern_tile_shfl(int* sum, int* input, int n) {
    int single_thread_sum = thread_sum(input, n);
    auto tile = tiled_partition<tilesize>(this_thread_block());
    int tile_sum = reduce_sum_tile_shfl<tilesize>(tile, single_thread_sum);
    if (tile.thread_rank() == 0) atomicAdd(sum, tile_sum);
}

void check_sum_kernel_block() {
    int n = 1 << 24;
    int blockSize = 256;
    int nBlocks = (n + blockSize - 1) / blockSize;
    int sharedBytes = blockSize * sizeof(int);
    int* hostData = (int*)malloc(n * sizeof(int));
    std::iota(hostData, hostData + n, 1);
    int* deviceData = nullptr;
    hipMalloc((void**)&deviceData, (n + 4) * sizeof(int));
    hipMemset(deviceData, 0, sizeof(int));
    // WARN: use deviceData[0] store sum result, 
    // but considering the alignment requirement of int4, we use deviceData + 4 as the beginning address of input
    // odd thing may happen is alignment is not satisfied
    hipMemcpy((char*)(deviceData + 4), hostData, n * sizeof(int), hipMemcpyHostToDevice);
    
    //sum_kernel_block<<<nBlocks, blockSize, sharedBytes>>>(deviceData, deviceData + 4, n);
    //sum_kernel_by_tile32<<<nBlocks, blockSize, sharedBytes>>>(deviceData, deviceData + 4, n);
    sum_kern_tile_shfl<32><<<nBlocks, blockSize>>>(deviceData, deviceData + 4, n);

    int hostResult;
    hipMemcpy((char*)&hostResult, (char*)deviceData, sizeof(int), hipMemcpyDeviceToHost);

    printf("result: %d, %d\n", hostResult, std::accumulate(hostData, hostData + n, 0));
    hipFree(deviceData);
}

// mock a global kernel calling the device reduce_sum function
__global__ void sum_kernel_block2(int* result, int* input, int n) {
    int idx = threadIdx.x;
    extern __shared__ int temp[];
    int mysum = 0;
    if (idx < n) {
        auto g = this_thread_block();
        mysum = reduce_sum(g, temp, input[idx]);
        printf("sum result, thread idx %d, %d \n", threadIdx.x, mysum);
    }
    if (idx == 0) *result = mysum;
}

void check_reduce_sum() {
    int datas[16] = {1,2,3,4,5,6,7,8,9,10,11,12,13,14,15,16};
    int* gdatas = nullptr;
    hipMalloc((void**)&gdatas, sizeof(datas));
    hipMemcpy((char*)gdatas, (char*)datas, sizeof(datas), hipMemcpyHostToDevice);
    int* result = (int*)malloc(sizeof(datas));
    const int size = 13;
    sum_kernel_block2<<<1, size, size * sizeof(int)>>>(gdatas, gdatas, size);
    hipMemcpy((char*)result, (char*)gdatas, sizeof(datas), hipMemcpyDeviceToHost);
    std::cout << result[0] << std::endl;
    hipFree(gdatas);
    
}
/*===============================================================================================*/



int main() {
    test_reduce_sum();
    //check_sum_kernel_block();
}   
